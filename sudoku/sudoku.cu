#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "util.h"

// The width and height of a sudoku board
#define BOARD_DIM 9

// The width and heigh of a square group in a sudoku board
#define GROUP_DIM 3

// The number of boards to pass to the solver at one time
#define BATCH_SIZE 128

/**
 * A board is an array of 81 cells. Each cell is encoded as a 16-bit integer.
 * Read about this encoding in the documentation for the digit_to_cell and
 * cell_to_digit functions' documentation.
 *
 * Boards are stored as a one-dimensional array. It doesn't matter if you use
 * row-major or column-major form (that just corresponds to a rotation of the
 * sudoku board) but you will need to convert column and row to a single index
 * when accessing the board to propagate constraints.
 */
typedef struct board {
  uint16_t cells[BOARD_DIM * BOARD_DIM];
} board_t;

// Declare a few functions. Documentation is with the function definition.
void print_board(board_t* board);
__host__ __device__ uint16_t digit_to_cell(int digit);
__host__ __device__ int cell_to_digit(uint16_t cell);

/**
 * Take an array of boards and solve them all. The number of boards will be no
 * more than BATCH_SIZE, but may be less if the total number of input boards
 * is not evenly-divisible by BATCH_SIZE.
 *
 * TODO: Implement this function! You will need to add a GPU kernel, and you
 *       will almost certainly want to write helper functions; that is fine.
 *       However, you should not modify any other functions in this file.
 *
 * \param boards      An array of boards that should be solved.
 * \param num_boards  The numebr of boards in the boards array
 */
void solve_boards(board_t* boards, size_t num_boards) {
  // TODO: Implement me!
}

/**
 * Take as input an integer value 0-9 (inclusive) and convert it to the encoded
 * cell form used for solving the sudoku. This encoding uses bits 1-9 to 
 * indicate which values may appear in this cell.
 *
 * For example, if bit 3 is set to 1, then the cell may hold a three. Cells that
 * have multiple possible values will have multiple bits set.
 *
 * The input digit 0 is treated specially. This value indicates a blank cell,
 * where any value from one to nine is possible.
 *
 * \param digit   An integer value 0-9 inclusive
 * \returns       The encoded form of digit using bits to indicate which values
 *                may appear in this cell.
 */
__host__ __device__ uint16_t digit_to_cell(int digit) {
  if(digit == 0) {
    // A zero indicates a blank cell. Numbers 1-9 are possible, so set bits 1-9.
    return 0x3FE;
  } else {
    // Otherwise we have a fixed value. Set the corresponding bit in the board.
    return 1<<digit;
  }
}

/*
 * Convert an encoded cell back to its digit form. A cell with two or more
 * possible values will be encoded as a zero. Cells with one possible value
 * will be converted to that value.
 *
 * For example, if the provided cell has only bit three set, this function will
 * return the value 3.
 *
 * \param cell  An encoded cell that uses bits to indicate which values could
 *              appear at this point in the board.
 * \returns     The value that must appear in the cell if there is only one
 *              possibility, or zero otherwise.
 */
__host__ __device__ int cell_to_digit(uint16_t cell) {
  // Get the index of the least-significant bit in this cell's value
#if defined(__CUDA_ARCH__)
  int msb = __clz(cell);
  int lsb = sizeof(unsigned int)*8 - msb - 1;
#else
  int lsb = __builtin_ctz(cell);
#endif

  // Is there only one possible value for this cell? If so, return it.
  // Otherwise return zero.
  if(cell == 1<<lsb) return lsb;
  else return 0;
}

/**
 * Read in a sudoku board from a string. Boards are represented as an array of
 * 81 16-bit integers. Each integer corresponds to a cell in the board. Bits
 * 1-9 of the integer indicate whether the values 1, 2, ..., 8, or 9 could
 * appear in the given cell. A zero in the input indicates a blank cell, where
 * any value could appear.
 *
 * \param output  The location where the board will be written
 * \param str     The input string that encodes the board
 * \returns       true if parsing succeeds, false otherwise
 */
bool read_board(board_t* output, const char* str) {
  for(int index=0; index<BOARD_DIM*BOARD_DIM; index++) {
    if(str[index] < '0' || str[index] > '9') return false;

    // Convert the character value to an equivalent integer
    int value = str[index] - '0';

    // Set the value in the board
    output->cells[index] = digit_to_cell(value);
  }

  return true;
}

/**
 * Print a sudoku board. Any cell with a single possible value is printed. All
 * cells with two or more possible values are printed as blanks.
 *
 * \param board   The sudoku board to print
 */
void print_board(board_t* board) {
  for(int row=0; row<BOARD_DIM; row++) {
    // Print horizontal dividers
    if(row != 0 && row % GROUP_DIM == 0) {
      for(int col=0; col<BOARD_DIM*2+BOARD_DIM/GROUP_DIM; col++) {
        printf("-");
      }
      printf("\n");
    }

    for(int col=0; col<BOARD_DIM; col++) {
      // Print vertical dividers
      if(col != 0 && col % GROUP_DIM == 0) printf("| ");

      // Compute the index of this cell in the board array
      int index = col + row * BOARD_DIM;

      // Get the index of the least-significant bit in this cell's value
      int digit = cell_to_digit(board->cells[index]);

      // Print the digit if it's not a zero. Otherwise print a blank.
      if(digit != 0) printf("%d ", digit);
      else printf("  ");
    }
    printf("\n");
  }
  printf("\n");
}

/**
 * Check through a batch of boards to see how many were solved correctly.
 *
 * \param boards        An array of (hopefully) solved boards
 * \param solutions     An array of solution boards
 * \param num_boards    The number of boards and solutions
 * \param solved_count  Output: A pointer to the count of solved boards.
 * \param error:count   Output: A pointer to the count of incorrect boards.
 */
void check_solutions(board_t* boards, board_t* solutions, size_t num_boards,
    size_t* solved_count, size_t* error_count) {

  // Loop over all the boards in this batch
  for(int i=0; i<num_boards; i++) {
    // Does the board match the solution?
    if(memcmp(&boards[i], &solutions[i], sizeof(board_t)) == 0) {
      // Yes. Record a solved board
      (*solved_count)++;
    } else {
      // No. Make sure the board doesn't have any constraints that rule out
      // values that are supposed to appear in the solution.
      bool valid = true;
      for(int j=0; j<BOARD_DIM * BOARD_DIM; j++) {
        if((boards[i].cells[j] & solutions[i].cells[j]) == 0) {
          valid = false;
        }
      }

      // If the board contains an incorrect constraint, record an error
      if(!valid) (*error_count)++;
    }
  }
}

/**
 * Entry point for the program
 */
int main(int argc, char** argv) {
  // Check arguments
  if(argc != 2) {
    fprintf(stderr, "Usage: %s <input file name>\n", argv[0]);
    exit(1);
  }

  // Try to open the input file
  FILE* input = fopen(argv[1], "r");
  if(input == NULL) {
    fprintf(stderr, "Failed to open input file %s.\n", argv[1]);
    perror(NULL);
    exit(2);
  }

  // Keep track of total boards, boards solved, and incorrect outputs
  size_t board_count = 0;
  size_t solved_count = 0;
  size_t error_count = 0;

  // Keep track of time spent solving
  size_t solving_time = 0;

  // Reserve space for a batch of boards and solutions
  board_t boards[BATCH_SIZE];
  board_t solutions[BATCH_SIZE];

  // Keep track of how many boards we've read in this batch
  size_t batch_count = 0;

  // Read the input file line-by-line
  char* line = NULL;
  size_t line_capacity = 0;
  while(getline(&line, &line_capacity, input) > 0) {
    // Read in the starting board
    if(!read_board(&boards[batch_count], line)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Read in the solution board
    if(!read_board(&solutions[batch_count], line + BOARD_DIM * BOARD_DIM + 1)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Move to the next index in the batch
    batch_count++;

    // Also increment the total count of boards
    board_count++;

    // If we finished a batch, run the solver
    if(batch_count == BATCH_SIZE) {
      size_t start_time = time_ms();
      solve_boards(boards, batch_count);
      solving_time += time_ms() - start_time;

      check_solutions(boards, solutions, batch_count,
          &solved_count, &error_count);

      // Reset the batch count
      batch_count = 0;
    }
  }

  // Check if there's an incomplete batch to solve
  if(batch_count > 0) {
    size_t start_time = time_ms();
    solve_boards(boards, batch_count);
    solving_time += time_ms() - start_time;

    check_solutions(boards, solutions, batch_count, &solved_count,
        &error_count);
  }

  // Print stats
  double seconds = (double)solving_time / 1000;
  double solving_rate = (double)solved_count / seconds;
  
  // Don't print nan when solver is not implemented
  if(seconds < 0.01) solving_rate = 0;

  printf("Boards: %lu\n", board_count);
  printf("Boards Solved: %lu\n", solved_count);
  printf("Errors: %lu\n", error_count);
  printf("Total Solving Time: %lums\n", solving_time);
  printf("Solving Rate: %.2f sudoku/second\n", solving_rate);

  return 0;
}

